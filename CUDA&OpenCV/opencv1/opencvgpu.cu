#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include"opencvgpu.h"
__global__ void bitwise_image_kernel(cuda::PtrStepSz<uchar3> d_src,cuda::PtrStepSz<uchar3> d_dst){
	unsigned int x = blockDim.x*blockIdx.x+threadIdx.x;
	unsigned int y = blockDim.y*blockIdx.y+threadIdx.y;
	if (x<d_src.cols&&y<d_src.rows){
		d_dst(y,x).x = 255-d_src(y,x).x;
		d_dst(y,x).y = 255-d_src(y,x).y;
		d_dst(y,x).z = 255-d_src(y,x).z;
	}
}

void bitwise_image(cuda::GpuMat src,cuda::GpuMat dst){
	int uint = 32;
	dim3 thread(uint,uint);
	dim3 block((src.cols+uint-1)/uint,(src.rows+uint-1)/uint);
	bitwise_image_kernel<<<block,thread>>>(src,dst);
}
